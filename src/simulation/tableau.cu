#include "simulation/tableau.cuh"
#include "core/types.hpp"
#include <hip/hip_runtime.h>

namespace RKF78
{
    __constant__ Float d_c[NStages - 1];
    __constant__ Float d_b[NStages];
    __constant__ Float d_be[NStages];
    __constant__ Float d_a[NStages - 1][NStages - 1];

    hipError_t initialize_device_tableau()
    {
        hipError_t err;
        err = hipMemcpyToSymbol(HIP_SYMBOL(d_c), host::c, sizeof(host::c));
        if (err != hipSuccess)
        {
            return err;
        }
        err = hipMemcpyToSymbol(HIP_SYMBOL(d_b), host::b, sizeof(host::b));
        if (err != hipSuccess)
        {
            return err;
        }
        err = hipMemcpyToSymbol(HIP_SYMBOL(d_be), host::be, sizeof(host::be));
        if (err != hipSuccess)
        {
            return err;
        }
        err = hipMemcpyToSymbol(HIP_SYMBOL(d_a), host::a, sizeof(host::a));
        return err;
    }
}